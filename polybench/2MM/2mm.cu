#include "hip/hip_runtime.h"
/**
 * 2mm.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "../polybenchUtilFuncts.h"

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05
#define GPU_DEVICE 0

/* Problem size. */
// # define NI 2048
// # define NJ 2048
// # define NK 2048
// # define NL 2048

/* Thread block dimensions */
// #define DIM_THREAD_BLOCK_X 32
#define DIM_THREAD_BLOCK_Y 8

/* Can switch DATA_TYPE between float and double */
typedef float DATA_TYPE;

void init_array(DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE* D, int ni, int nj, int nk, int nl) // [MODIFIED CODE] // void init_array(DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE* D)
{
	int i, j;

	for (i = 0; i < ni; i++)
	{
		for (j = 0; j < nk; j++)
		{
			A[i*nk + j] = ((DATA_TYPE) i*j) / ni; // [MODIFIED CODE] // A[i*NI + j]
		}
	}

	for (i = 0; i < nk; i++)
	{
		for (j = 0; j < nj; j++)
		{
			B[i*nj + j] = ((DATA_TYPE) i*(j+1)) / nj; // [MODIFIED CODE]
		}
	}

	for (i = 0; i < nl; i++)
	{
		for (j = 0; j < nj; j++)
		{
			C[i*nj + j] = ((DATA_TYPE) i*(j+3)) / nl; // [MODIFIED CODE]
		}
	}

	for (i = 0; i < ni; i++)
	{
		for (j = 0; j < nl; j++)
		{
			D[i*nl + j] = ((DATA_TYPE) i*(j+2)) / nk; // [MODIFIED CODE]
		}
	}
}

void compareResults(DATA_TYPE *E, DATA_TYPE *E_outputFromGpu, int ni, int nl) // [MODIFIED CODE] // void compareResults(DATA_TYPE *E, DATA_TYPE *E_outputFromGpu)
{
	int i,j,fail;
	fail = 0;

	for (i = 0; i < ni; i++)
	{
		for (j = 0; j < nl; j++)
		{
			if (percentDiff(E[i*nl + j], E_outputFromGpu[i*nl + j]) > PERCENT_DIFF_ERROR_THRESHOLD) // [MODIFIED CODE]
			{
				fail++;
			}
		}
	}
	// print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}

void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n", GPU_DEVICE, deviceProp.name);
	hipSetDevice(GPU_DEVICE);
}

__global__ void mm2_kernel1(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *C, int ni, int nj, int nk) // [MODIFIED CODE]
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < ni) && (j < nj))
	{ 
		int k;
		for (k = 0; k < nk; k++)
		{
			C[i * nj + j] += A[i * nk + k] * B[k * nj + j]; // [MODIFIED CODE]
		}
	}
}

__global__ void mm2_kernel2(DATA_TYPE *C, DATA_TYPE *D, DATA_TYPE *E, int ni, int nj, int nl) // [MODIFIED CODE]
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < ni) && (j < nl))
	{ 
		int k;
		for (k = 0; k < nj; k++)
		{
			E[i * nl + j] += C[i * nj + k] * D[k * nl + j]; // [MODIFIED CODE]
		}
	}
}

void mm2_cpu(DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE* D, DATA_TYPE* E, int ni, int nj, int nk, int nl) // [MODIFIED CODE]
{
	int i, j, k;

	for (i = 0; i < ni; i++)
	{
		for (j = 0; j < nj; j++)
		{
			C[i*nj + j] = 0.0;
			for (k = 0; k < nk; ++k)
			{
				C[i*nj + j] += A[i*nk + k] * B[k*nj + j];
			}
		}
	}

	for (i = 0; i < ni; i++)
	{
		for (j = 0; j < nl; j++)
		{
			E[i*nl + j] = 0.0;
			for (k = 0; k < nj; ++k)
			{
				E[i*nl + j] += C[i*nj + k] * D[k*nl + j];
			}
		}
	}
}

void mm2Cuda(DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE* D, DATA_TYPE* E, DATA_TYPE* E_outputFromGpu, int ni, int nj, int nk, int nl, int dim_thread_block_x) // [MODIFIED CODE]
{
	double t_start, t_end;

	DATA_TYPE *A_gpu, *B_gpu, *C_gpu, *D_gpu, *E_gpu;

	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * ni * nk);
	hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * nk * nj);
	hipMalloc((void **)&C_gpu, sizeof(DATA_TYPE) * ni * nj);
	hipMalloc((void **)&D_gpu, sizeof(DATA_TYPE) * nj * nl);
	hipMalloc((void **)&E_gpu, sizeof(DATA_TYPE) * ni * nl);

	hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * ni * nk, hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * nk * nj, hipMemcpyHostToDevice);
	hipMemcpy(C_gpu, C, sizeof(DATA_TYPE) * ni * nj, hipMemcpyHostToDevice);
	hipMemcpy(D_gpu, D, sizeof(DATA_TYPE) * nj * nl, hipMemcpyHostToDevice);
	hipMemcpy(E_gpu, E, sizeof(DATA_TYPE) * ni * nl, hipMemcpyHostToDevice);	

	dim3 block(dim_thread_block_x, DIM_THREAD_BLOCK_Y);
	dim3 grid1((size_t)ceil( ((float)nj) / ((float)block.x) ), (size_t)ceil( ((float)ni) / ((float)block.y)) );
	dim3 grid2((size_t)ceil( ((float)nl) / ((float)block.x) ), (size_t)ceil( ((float)ni) / ((float)block.y)) );

	t_start = rtclock();
	mm2_kernel1<<<grid1, block>>>(A_gpu, B_gpu, C_gpu, ni, nj, nk); // [MODIFIED CODE]
	hipDeviceSynchronize();
	mm2_kernel2<<<grid2, block>>>(C_gpu, D_gpu, E_gpu, ni, nj, nl); // [MODIFIED CODE]
	hipDeviceSynchronize();
	t_end = rtclock();
	fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);

	hipMemcpy(E_outputFromGpu, E_gpu, sizeof(DATA_TYPE) * ni * nl, hipMemcpyDeviceToHost);

	hipFree(A_gpu); hipFree(B_gpu); hipFree(C_gpu); hipFree(D_gpu); hipFree(E_gpu);
}

int main(int argc, char** argv)
{
	double t_start, t_end;

	int dim_thread_block_x = 32;
	int size = 32; //2048; // [MODIFIED CODE]
	int ni = size, nj = size, nk = size, nl = size; // [MODIFIED CODE]

	for (int i = 1; i < argc; i++) {
		if (!strcmp(argv[i], "-size") && i + 1 < argc) {
			size = atoi(argv[++i]);
			ni = nj = nk = nl = size;
		}
		if (!strcmp(argv[i], "-blockDimX") && i + 1 < argc) {
			dim_thread_block_x = atoi(argv[++i]);
		}
		if (size < dim_thread_block_x || size < DIM_THREAD_BLOCK_Y) {
			fprintf(stderr, "Error: size must be >= dim_thread_block_x=%d and dim_thread_block_y=%d.\n", dim_thread_block_x, DIM_THREAD_BLOCK_Y);
			exit(1);
		}
	}
	printf("size=%d, dim_thread_block_x=%d\n", size, dim_thread_block_x);

	DATA_TYPE *C = (DATA_TYPE*)malloc(ni*nj*sizeof(DATA_TYPE));
	DATA_TYPE *A = (DATA_TYPE*)malloc(ni*nk*sizeof(DATA_TYPE));
	DATA_TYPE *B = (DATA_TYPE*)malloc(nk*nj*sizeof(DATA_TYPE));
	DATA_TYPE *D = (DATA_TYPE*)malloc(nj*nl*sizeof(DATA_TYPE));
	DATA_TYPE *E = (DATA_TYPE*)malloc(ni*nl*sizeof(DATA_TYPE));
	DATA_TYPE *E_outputFromGpu = (DATA_TYPE*)malloc(ni*nl*sizeof(DATA_TYPE));

	init_array(A, B, C, D, ni, nj, nk, nl);
	GPU_argv_init();

	mm2Cuda(A, B, C, D, E, E_outputFromGpu, ni, nj, nk, nl, dim_thread_block_x); // [MODIFIED CODE]

	// t_start = rtclock();
	// mm2_cpu(A, B, C, D, E, ni, nj, nk, nl);
	// t_end = rtclock();
	// fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);

	// compareResults(E, E_outputFromGpu, ni, nl);

	free(C);
	free(A);
	free(B);
	free(D);
	free(E);
	free(E_outputFromGpu);

	return 0;
}
 